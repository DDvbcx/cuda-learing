#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <>
#define BLOCK_DIM 1024
#define max_function(a, b) ((a) > (b) ? (a) : (b))

double get_walltime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)(tp.tv_sec + tp.tv_usec * 1e-6)
}

// Kernel for expanding the input tensor to match the output tensor
__global__ void _expand_kernel(float *input, float *output, int nDims,
                               int outputsize, int *inputShape,
                               int *outputShape) {
    int outputIdx = blockIdx.x * blockDim.x + threadIdx.x; // Linear index for output array

    if (outputIdx < outputsize) { // Boundary check
        int inputIdx = 0;  // Index in the input array
        int temp = 1;      // Cumulative multiplier for input indices
        int tmp = 1;       // Temporary value for current dimension index
        int v = outputIdx; // Current output index to process

        for (int i = nDims - 1; i >= 0; --i) {
            if (i == 0) {
                tmp = v; // Handle the first dimension
            } else {
                tmp = v % outputShape[i]; // Calculate dimension index
            }

            if (inputShape[i] == 1) {
                // Broadcasting: input dimension is 1, so index is always 0
                inputIdx += 0;
            } else {
                // Map the current output index to input index
                inputIdx += tmp * temp;
            }

            temp *= inputShape[i];
            v = v / outputShape[i];
        }

        // Assign the input value to the corresponding output index
        output[outputIdx] = input[inputIdx];
    }
}

// Host function to launch the kernel and handle memory management
void expand(float *cpu_input, float *cpu_output, int nDims, int inputsize,
            int outputsize, int *cpu_inputShape, int *cpu_outputShape) {
    double st, ela;
    st = get_walltime();

    // Define grid and block dimensions
    int num_blocks = ceil(outputsize / (double)BLOCK_DIM);
    dim3 block_dim(BLOCK_DIM, 1, 1);
    dim3 grid_dim(num_blocks, 1, 1);

    // Allocate memory on the device
    float *input, *output;
    hipMalloc((void **)&input, inputsize * sizeof(float));
    hipMalloc((void **)&output, outputsize * sizeof(float));
    hipMemcpy(input, cpu_input, inputsize * sizeof(float), hipMemcpyHostToDevice);

    int *inputShape, *outputShape;
    hipMalloc((void **)&inputShape, nDims * sizeof(int));
    hipMalloc((void **)&outputShape, nDims * sizeof(int));
    hipMemcpy(inputShape, cpu_inputShape, nDims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(outputShape, cpu_outputShape, nDims * sizeof(int), hipMemcpyHostToDevice);

    // Record kernel execution time
    hipEvent_t start, stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Launch the kernel
    _expand_kernel<<<grid_dim, block_dim>>>(input, output, nDims, outputsize,
                                            inputShape, outputShape);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop);

    // Copy the results back to the host
    hipMemcpy(cpu_output, output, outputsize * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(input);
    hipFree(output);
    hipFree(inputShape);
    hipFree(outputShape);

    ela = get_walltime() - st;

    printf("kernel time: %.4f s, total time: %.4f s\n", ker_time / 1000., ela);
}

// Main function
int main() {
    float *cpu_input, *cpu_output;
    int nDims = 4;
    int cpu_inputShape[] = {2, 1, 1, 2};
    int cpu_outputShape[] = {2, 2, 2, 2};

    int inputsize = 1, outputsize = 1;
    for (int i = 0; i < nDims; i++) {
        inputsize *= cpu_inputShape[i];
        outputsize *= cpu_outputShape[i];
    }

    // Allocate host memory
    cpu_input = (float *)malloc(inputsize * sizeof(float));
    cpu_output = (float *)malloc(outputsize * sizeof(float));

    // Initialize the input array
    for (int i = 0; i < inputsize; i++) {
        cpu_input[i] = i;
    }

    // Perform the expand operation
    expand(cpu_input, cpu_output, nDims, inputsize, outputsize, cpu_inputShape, cpu_outputShape);

    // Verify the results
    float result[] = {0, 1, 0, 1, 0, 1, 0, 1, 2, 3, 2, 3, 2, 3, 2, 3};
    float error = 0;
    for (int i = 0; i < outputsize; i++) {
        error = fmax(error, fabs(result[i] - cpu_output[i]));
    }

    printf("Max error: %.4e\n", error);

    // Free host memory
    free(cpu_input);
    free(cpu_output);

    return 0;
}
