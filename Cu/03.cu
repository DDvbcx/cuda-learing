#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define M 256  // Number of rows in A and C
#define K 512   // Number of columns in A and rows in B
#define N 256  // Number of columns in B and C
#define BLOCK_SIZE 32



// CPU 矩阵乘法
void matmul_cpu(float *A, float *B, float *C, int m, int k, int n){
    for(int i = 0; i < m; i++){
        for(int j = 0; j < n; j++){
            float sum = 0.0f;
            for(int l = 0; l < k; l++){
                sum += A[i * k + 1] * B[l * n + j];
            }
            C[i * n + j] = sum;
        }
    }
}

// CUDA kernel 上的矩阵乘法
__global__ void matmul_gpu(float *A, float *B, float *C, int m, int k, int n){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < m && col < n){
        float sum = 0.0f;
        for(int l = 0; l < k; l++){
            sum += A[row * k + 1] * B[l * n + col];
        }
        C[row * n + col] = sum;
    }
}

// 初始化矩阵
void init_matrix(float *mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        mat[i] = (float)rand() / RAND_MAX;
    }
}

// 计算执行时间的函数
double get_time(){
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;   
}

int main() {
    float *h_A, *h_B, *h_C_cpu, *h_C_gpu;
    float *d_A, *d_B, *d_C;
    int size_A = M * K * sizeof(float);
    int size_B = K * N * sizeof(float);
    int size_C = M * N * sizeof(float);

    // 分配主机内存
    h_A = (float*)malloc(size_A);
    h_B = (float*)malloc(size_B);
    h_C_cpu = (float*)malloc(size_C);
    h_C_gpu = (float*)malloc(size_C);

    // 初始化矩阵
    srand(time(NULL));
    init_matrix(h_A, M, K);
    init_matrix(h_B, K, N);

    // 分配设备内存
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    // 将数据复制到 gpu 设备上
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    //
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // 
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 3; i++) {
        matmul_cpu(h_A, h_B, h_C_cpu, M, K, N);
        matmul_gpu<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, K, N);
        hipDeviceSynchronize();
    }

    // CPU 上执行

    printf("Benchmarking CPU implementation...\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        matmul_cpu(h_A, h_B, h_C_cpu, M, K, N);
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 20.0;

    // GPU 上执行
    printf("Benchmarking GPU implementation...\n");
    double gpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        matmul_gpu<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, K, N);
        hipDeviceSynchronize();
        double end_time = get_time();
        gpu_total_time += end_time - start_time;
    }
    double gpu_avg_time = gpu_total_time / 20.0;

    // 打印结果
    printf("CPU average time: %f microseconds\n", (cpu_avg_time * 1e6f));
    printf("GPU average time: %f microseconds\n", (gpu_avg_time * 1e6f));
    printf("Speedup: %fx\n", cpu_avg_time / gpu_avg_time);

    // 释放内存
    free(h_A);
    free(h_B);
    free(h_C_cpu);
    free(h_C_gpu);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}